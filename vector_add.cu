
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel for vector addition
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    // Calculate the global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1 << 20; // Size of the vectors (1M elements)
    size_t bytes = N * sizeof(float);

    // Allocate memory on the host (CPU)
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize vectors with some values
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    // Launch the kernel
    vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
